#include "hip/hip_runtime.h"




<!DOCTYPE html>
<html>
<head>
 <meta http-equiv="Content-Type" content="text/html; charset=UTF-8" >
 <meta http-equiv="X-UA-Compatible" content="IE=edge,chrome=1" >
 
 <meta name="ROBOTS" content="NOARCHIVE">
 
 <link rel="icon" type="image/vnd.microsoft.icon" href="https://ssl.gstatic.com/codesite/ph/images/phosting.ico">
 
 
 <script type="text/javascript">
 
 
 
 
 var codesite_token = "MBDwIWyWnVeH7W7374m1cEh0iI0:1371808304867";
 
 
 var CS_env = {"domainName":null,"assetHostPath":"https://ssl.gstatic.com/codesite/ph","profileUrl":"/u/116699586124044253698/","token":"MBDwIWyWnVeH7W7374m1cEh0iI0:1371808304867","relativeBaseUrl":"","projectName":"stanford-cs193g-sp2010","loggedInUserEmail":"gaurav.sachin007@gmail.com","assetVersionPath":"https://ssl.gstatic.com/codesite/ph/18376132045800511552","projectHomeUrl":"/p/stanford-cs193g-sp2010"};
 var _gaq = _gaq || [];
 _gaq.push(
 ['siteTracker._setAccount', 'UA-18071-1'],
 ['siteTracker._trackPageview']);
 
 (function() {
 var ga = document.createElement('script'); ga.type = 'text/javascript'; ga.async = true;
 ga.src = ('https:' == document.location.protocol ? 'https://ssl' : 'http://www') + '.google-analytics.com/ga.js';
 (document.getElementsByTagName('head')[0] || document.getElementsByTagName('body')[0]).appendChild(ga);
 })();
 
 </script>
 
 
 <title>block_scan.cu - 
 stanford-cs193g-sp2010 -
 
 
 Programming Massively Parallel Processors with CUDA - Google Project Hosting
 </title>
 <link type="text/css" rel="stylesheet" href="https://ssl.gstatic.com/codesite/ph/18376132045800511552/css/core.css">
 
 <link type="text/css" rel="stylesheet" href="https://ssl.gstatic.com/codesite/ph/18376132045800511552/css/ph_detail.css" >
 
 
 <link type="text/css" rel="stylesheet" href="https://ssl.gstatic.com/codesite/ph/18376132045800511552/css/d_sb.css" >
 
 
 
<!--[if IE]>
 <link type="text/css" rel="stylesheet" href="https://ssl.gstatic.com/codesite/ph/18376132045800511552/css/d_ie.css" >
<![endif]-->
 <style type="text/css">
 .menuIcon.off { background: no-repeat url(https://ssl.gstatic.com/codesite/ph/images/dropdown_sprite.gif) 0 -42px }
 .menuIcon.on { background: no-repeat url(https://ssl.gstatic.com/codesite/ph/images/dropdown_sprite.gif) 0 -28px }
 .menuIcon.down { background: no-repeat url(https://ssl.gstatic.com/codesite/ph/images/dropdown_sprite.gif) 0 0; }
 
 
 
  tr.inline_comment {
 background: #fff;
 vertical-align: top;
 }
 div.draft, div.published {
 padding: .3em;
 border: 1px solid #999; 
 margin-bottom: .1em;
 font-family: arial, sans-serif;
 max-width: 60em;
 }
 div.draft {
 background: #ffa;
 } 
 div.published {
 background: #e5ecf9;
 }
 div.published .body, div.draft .body {
 padding: .5em .1em .1em .1em;
 max-width: 60em;
 white-space: pre-wrap;
 white-space: -moz-pre-wrap;
 white-space: -pre-wrap;
 white-space: -o-pre-wrap;
 word-wrap: break-word;
 font-size: 1em;
 }
 div.draft .actions {
 margin-left: 1em;
 font-size: 90%;
 }
 div.draft form {
 padding: .5em .5em .5em 0;
 }
 div.draft textarea, div.published textarea {
 width: 95%;
 height: 10em;
 font-family: arial, sans-serif;
 margin-bottom: .5em;
 }

 
 .nocursor, .nocursor td, .cursor_hidden, .cursor_hidden td {
 background-color: white;
 height: 2px;
 }
 .cursor, .cursor td {
 background-color: darkblue;
 height: 2px;
 display: '';
 }
 
 
.list {
 border: 1px solid white;
 border-bottom: 0;
}

 
 </style>
</head>
<body class="t4">
<script type="text/javascript">
 window.___gcfg = {lang: 'en'};
 (function() 
 {var po = document.createElement("script");
 po.type = "text/javascript"; po.async = true;po.src = "https://apis.google.com/js/plusone.js";
 var s = document.getElementsByTagName("script")[0];
 s.parentNode.insertBefore(po, s);
 })();
</script>
<div class="headbg">

 <div id="gaia">
 

 <span>
 
 
 
 <a href="#" id="multilogin-dropdown" onclick="return false;"
 ><u><b>gaurav.sachin007@gmail.com</b></u> <small>&#9660;</small></a>
 
 
 | <a href="/u/116699586124044253698/" id="projects-dropdown" onclick="return false;"
 ><u>My favorites</u> <small>&#9660;</small></a>
 | <a href="/u/116699586124044253698/" onclick="_CS_click('/gb/ph/profile');"
 title="Profile, Updates, and Settings"
 ><u>Profile</u></a>
 | <a href="https://www.google.com/accounts/Logout?continue=https%3A%2F%2Fcode.google.com%2Fp%2Fstanford-cs193g-sp2010%2Fsource%2Fbrowse%2Ftrunk%2Ftutorials%2Fblock_scan.cu" 
 onclick="_CS_click('/gb/ph/signout');"
 ><u>Sign out</u></a>
 
 </span>

 </div>

 <div class="gbh" style="left: 0pt;"></div>
 <div class="gbh" style="right: 0pt;"></div>
 
 
 <div style="height: 1px"></div>
<!--[if lte IE 7]>
<div style="text-align:center;">
Your version of Internet Explorer is not supported. Try a browser that
contributes to open source, such as <a href="http://www.firefox.com">Firefox</a>,
<a href="http://www.google.com/chrome">Google Chrome</a>, or
<a href="http://code.google.com/chrome/chromeframe/">Google Chrome Frame</a>.
</div>
<![endif]-->



 <table style="padding:0px; margin: 0px 0px 10px 0px; width:100%" cellpadding="0" cellspacing="0"
 itemscope itemtype="http://schema.org/CreativeWork">
 <tr style="height: 58px;">
 
 
 
 <td id="plogo">
 <link itemprop="url" href="/p/stanford-cs193g-sp2010">
 <a href="/p/stanford-cs193g-sp2010/">
 
 <img src="https://ssl.gstatic.com/codesite/ph/images/defaultlogo.png" alt="Logo" itemprop="image">
 
 </a>
 </td>
 
 <td style="padding-left: 0.5em">
 
 <div id="pname">
 <a href="/p/stanford-cs193g-sp2010/"><span itemprop="name">stanford-cs193g-sp2010</span></a>
 </div>
 
 <div id="psum">
 <a id="project_summary_link"
 href="/p/stanford-cs193g-sp2010/"><span itemprop="description">Programming Massively Parallel Processors with CUDA</span></a>
 
 </div>
 
 
 </td>
 <td style="white-space:nowrap;text-align:right; vertical-align:bottom;">
 
 <form action="/hosting/search">
 <input size="30" name="q" value="" type="text">
 
 <input type="submit" name="projectsearch" value="Search projects" >
 </form>
 
 </tr>
 </table>

</div>

 
<div id="mt" class="gtb"> 
 <a href="/p/stanford-cs193g-sp2010/" class="tab ">Project&nbsp;Home</a>
 
 
 
 
 
 
 <a href="/p/stanford-cs193g-sp2010/w/list" class="tab ">Wiki</a>
 
 
 
 
 
 <a href="/p/stanford-cs193g-sp2010/issues/list"
 class="tab ">Issues</a>
 
 
 
 
 
 <a href="/p/stanford-cs193g-sp2010/source/checkout"
 class="tab active">Source</a>
 
 
 
 
 
 
 
 
 <div class=gtbc></div>
</div>
<table cellspacing="0" cellpadding="0" width="100%" align="center" border="0" class="st">
 <tr>
 
 
 
 
 
 
 <td class="subt">
 <div class="st2">
 <div class="isf">
 
 


 <span class="inst1"><a href="/p/stanford-cs193g-sp2010/source/checkout">Checkout</a></span> &nbsp;
 <span class="inst2"><a href="/p/stanford-cs193g-sp2010/source/browse/">Browse</a></span> &nbsp;
 <span class="inst3"><a href="/p/stanford-cs193g-sp2010/source/list">Changes</a></span> &nbsp;
 
 
 
 
 
 
 
 </form>
 <script type="text/javascript">
 
 function codesearchQuery(form) {
 var query = document.getElementById('q').value;
 if (query) { form.action += '%20' + query; }
 }
 </script>
 </div>
</div>

 </td>
 
 
 
 <td align="right" valign="top" class="bevel-right"></td>
 </tr>
</table>


<script type="text/javascript">
 var cancelBubble = false;
 function _go(url) { document.location = url; }
</script>
<div id="maincol"
 
>

 




<div class="expand">
<div id="colcontrol">
<style type="text/css">
 #file_flipper { white-space: nowrap; padding-right: 2em; }
 #file_flipper.hidden { display: none; }
 #file_flipper .pagelink { color: #0000CC; text-decoration: underline; }
 #file_flipper #visiblefiles { padding-left: 0.5em; padding-right: 0.5em; }
</style>
<table id="nav_and_rev" class="list"
 cellpadding="0" cellspacing="0" width="100%">
 <tr>
 
 <td nowrap="nowrap" class="src_crumbs src_nav" width="33%">
 <strong class="src_nav">Source path:&nbsp;</strong>
 <span id="crumb_root">
 
 <a href="/p/stanford-cs193g-sp2010/source/browse/">svn</a>/&nbsp;</span>
 <span id="crumb_links" class="ifClosed"><a href="/p/stanford-cs193g-sp2010/source/browse/trunk/">trunk</a><span class="sp">/&nbsp;</span><a href="/p/stanford-cs193g-sp2010/source/browse/trunk/tutorials/">tutorials</a><span class="sp">/&nbsp;</span>block_scan.cu</span>
 
 


 </td>
 
 
 <td nowrap="nowrap" width="33%" align="center">
 <a href="/p/stanford-cs193g-sp2010/source/browse/trunk/tutorials/block_scan.cu?edit=1"
 ><img src="https://ssl.gstatic.com/codesite/ph/images/pencil-y14.png"
 class="edit_icon">Edit file</a>
 </td>
 
 
 <td nowrap="nowrap" width="33%" align="right">
 <table cellpadding="0" cellspacing="0" style="font-size: 100%"><tr>
 
 
 <td class="flipper"><b>r285</b></td>
 
 </tr></table>
 </td> 
 </tr>
</table>

<div class="fc">
 
 
 
<style type="text/css">
.undermouse span {
 background-image: url(https://ssl.gstatic.com/codesite/ph/images/comments.gif); }
</style>
<table class="opened" id="review_comment_area"
><tr>
<td id="nums">
<pre><table width="100%"><tr class="nocursor"><td></td></tr></table></pre>
<pre><table width="100%" id="nums_table_0"><tr id="gr_svn285_1"

><td id="1"><a href="#1">1</a></td></tr
><tr id="gr_svn285_2"

><td id="2"><a href="#2">2</a></td></tr
><tr id="gr_svn285_3"

><td id="3"><a href="#3">3</a></td></tr
><tr id="gr_svn285_4"

><td id="4"><a href="#4">4</a></td></tr
><tr id="gr_svn285_5"

><td id="5"><a href="#5">5</a></td></tr
><tr id="gr_svn285_6"

><td id="6"><a href="#6">6</a></td></tr
><tr id="gr_svn285_7"

><td id="7"><a href="#7">7</a></td></tr
><tr id="gr_svn285_8"

><td id="8"><a href="#8">8</a></td></tr
><tr id="gr_svn285_9"

><td id="9"><a href="#9">9</a></td></tr
><tr id="gr_svn285_10"

><td id="10"><a href="#10">10</a></td></tr
><tr id="gr_svn285_11"

><td id="11"><a href="#11">11</a></td></tr
><tr id="gr_svn285_12"

><td id="12"><a href="#12">12</a></td></tr
><tr id="gr_svn285_13"

><td id="13"><a href="#13">13</a></td></tr
><tr id="gr_svn285_14"

><td id="14"><a href="#14">14</a></td></tr
><tr id="gr_svn285_15"

><td id="15"><a href="#15">15</a></td></tr
><tr id="gr_svn285_16"

><td id="16"><a href="#16">16</a></td></tr
><tr id="gr_svn285_17"

><td id="17"><a href="#17">17</a></td></tr
><tr id="gr_svn285_18"

><td id="18"><a href="#18">18</a></td></tr
><tr id="gr_svn285_19"

><td id="19"><a href="#19">19</a></td></tr
><tr id="gr_svn285_20"

><td id="20"><a href="#20">20</a></td></tr
><tr id="gr_svn285_21"

><td id="21"><a href="#21">21</a></td></tr
><tr id="gr_svn285_22"

><td id="22"><a href="#22">22</a></td></tr
><tr id="gr_svn285_23"

><td id="23"><a href="#23">23</a></td></tr
><tr id="gr_svn285_24"

><td id="24"><a href="#24">24</a></td></tr
><tr id="gr_svn285_25"

><td id="25"><a href="#25">25</a></td></tr
><tr id="gr_svn285_26"

><td id="26"><a href="#26">26</a></td></tr
><tr id="gr_svn285_27"

><td id="27"><a href="#27">27</a></td></tr
><tr id="gr_svn285_28"

><td id="28"><a href="#28">28</a></td></tr
><tr id="gr_svn285_29"

><td id="29"><a href="#29">29</a></td></tr
><tr id="gr_svn285_30"

><td id="30"><a href="#30">30</a></td></tr
><tr id="gr_svn285_31"

><td id="31"><a href="#31">31</a></td></tr
><tr id="gr_svn285_32"

><td id="32"><a href="#32">32</a></td></tr
><tr id="gr_svn285_33"

><td id="33"><a href="#33">33</a></td></tr
><tr id="gr_svn285_34"

><td id="34"><a href="#34">34</a></td></tr
><tr id="gr_svn285_35"

><td id="35"><a href="#35">35</a></td></tr
><tr id="gr_svn285_36"

><td id="36"><a href="#36">36</a></td></tr
><tr id="gr_svn285_37"

><td id="37"><a href="#37">37</a></td></tr
><tr id="gr_svn285_38"

><td id="38"><a href="#38">38</a></td></tr
><tr id="gr_svn285_39"

><td id="39"><a href="#39">39</a></td></tr
><tr id="gr_svn285_40"

><td id="40"><a href="#40">40</a></td></tr
><tr id="gr_svn285_41"

><td id="41"><a href="#41">41</a></td></tr
><tr id="gr_svn285_42"

><td id="42"><a href="#42">42</a></td></tr
><tr id="gr_svn285_43"

><td id="43"><a href="#43">43</a></td></tr
><tr id="gr_svn285_44"

><td id="44"><a href="#44">44</a></td></tr
><tr id="gr_svn285_45"

><td id="45"><a href="#45">45</a></td></tr
><tr id="gr_svn285_46"

><td id="46"><a href="#46">46</a></td></tr
><tr id="gr_svn285_47"

><td id="47"><a href="#47">47</a></td></tr
><tr id="gr_svn285_48"

><td id="48"><a href="#48">48</a></td></tr
><tr id="gr_svn285_49"

><td id="49"><a href="#49">49</a></td></tr
><tr id="gr_svn285_50"

><td id="50"><a href="#50">50</a></td></tr
><tr id="gr_svn285_51"

><td id="51"><a href="#51">51</a></td></tr
><tr id="gr_svn285_52"

><td id="52"><a href="#52">52</a></td></tr
><tr id="gr_svn285_53"

><td id="53"><a href="#53">53</a></td></tr
><tr id="gr_svn285_54"

><td id="54"><a href="#54">54</a></td></tr
><tr id="gr_svn285_55"

><td id="55"><a href="#55">55</a></td></tr
><tr id="gr_svn285_56"

><td id="56"><a href="#56">56</a></td></tr
><tr id="gr_svn285_57"

><td id="57"><a href="#57">57</a></td></tr
><tr id="gr_svn285_58"

><td id="58"><a href="#58">58</a></td></tr
><tr id="gr_svn285_59"

><td id="59"><a href="#59">59</a></td></tr
><tr id="gr_svn285_60"

><td id="60"><a href="#60">60</a></td></tr
><tr id="gr_svn285_61"

><td id="61"><a href="#61">61</a></td></tr
><tr id="gr_svn285_62"

><td id="62"><a href="#62">62</a></td></tr
><tr id="gr_svn285_63"

><td id="63"><a href="#63">63</a></td></tr
><tr id="gr_svn285_64"

><td id="64"><a href="#64">64</a></td></tr
><tr id="gr_svn285_65"

><td id="65"><a href="#65">65</a></td></tr
><tr id="gr_svn285_66"

><td id="66"><a href="#66">66</a></td></tr
><tr id="gr_svn285_67"

><td id="67"><a href="#67">67</a></td></tr
><tr id="gr_svn285_68"

><td id="68"><a href="#68">68</a></td></tr
><tr id="gr_svn285_69"

><td id="69"><a href="#69">69</a></td></tr
><tr id="gr_svn285_70"

><td id="70"><a href="#70">70</a></td></tr
><tr id="gr_svn285_71"

><td id="71"><a href="#71">71</a></td></tr
><tr id="gr_svn285_72"

><td id="72"><a href="#72">72</a></td></tr
><tr id="gr_svn285_73"

><td id="73"><a href="#73">73</a></td></tr
><tr id="gr_svn285_74"

><td id="74"><a href="#74">74</a></td></tr
><tr id="gr_svn285_75"

><td id="75"><a href="#75">75</a></td></tr
><tr id="gr_svn285_76"

><td id="76"><a href="#76">76</a></td></tr
><tr id="gr_svn285_77"

><td id="77"><a href="#77">77</a></td></tr
><tr id="gr_svn285_78"

><td id="78"><a href="#78">78</a></td></tr
><tr id="gr_svn285_79"

><td id="79"><a href="#79">79</a></td></tr
><tr id="gr_svn285_80"

><td id="80"><a href="#80">80</a></td></tr
><tr id="gr_svn285_81"

><td id="81"><a href="#81">81</a></td></tr
><tr id="gr_svn285_82"

><td id="82"><a href="#82">82</a></td></tr
><tr id="gr_svn285_83"

><td id="83"><a href="#83">83</a></td></tr
><tr id="gr_svn285_84"

><td id="84"><a href="#84">84</a></td></tr
><tr id="gr_svn285_85"

><td id="85"><a href="#85">85</a></td></tr
><tr id="gr_svn285_86"

><td id="86"><a href="#86">86</a></td></tr
><tr id="gr_svn285_87"

><td id="87"><a href="#87">87</a></td></tr
><tr id="gr_svn285_88"

><td id="88"><a href="#88">88</a></td></tr
><tr id="gr_svn285_89"

><td id="89"><a href="#89">89</a></td></tr
><tr id="gr_svn285_90"

><td id="90"><a href="#90">90</a></td></tr
><tr id="gr_svn285_91"

><td id="91"><a href="#91">91</a></td></tr
><tr id="gr_svn285_92"

><td id="92"><a href="#92">92</a></td></tr
><tr id="gr_svn285_93"

><td id="93"><a href="#93">93</a></td></tr
><tr id="gr_svn285_94"

><td id="94"><a href="#94">94</a></td></tr
><tr id="gr_svn285_95"

><td id="95"><a href="#95">95</a></td></tr
><tr id="gr_svn285_96"

><td id="96"><a href="#96">96</a></td></tr
><tr id="gr_svn285_97"

><td id="97"><a href="#97">97</a></td></tr
></table></pre>
<pre><table width="100%"><tr class="nocursor"><td></td></tr></table></pre>
</td>
<td id="lines">
<pre><table width="100%"><tr class="cursor_stop cursor_hidden"><td></td></tr></table></pre>
<pre ><table id="src_table_0"><tr
id=sl_svn285_1

><td class="source">// This example demonstrates a block-wise inclusive<br></td></tr
><tr
id=sl_svn285_2

><td class="source">// parallel prefix sum (scan) algorithm.<br></td></tr
><tr
id=sl_svn285_3

><td class="source"><br></td></tr
><tr
id=sl_svn285_4

><td class="source">#include &lt;stdlib.h&gt;<br></td></tr
><tr
id=sl_svn285_5

><td class="source">#include &lt;stdio.h&gt;<br></td></tr
><tr
id=sl_svn285_6

><td class="source">#include &lt;vector&gt;<br></td></tr
><tr
id=sl_svn285_7

><td class="source">#include &lt;iostream&gt;<br></td></tr
><tr
id=sl_svn285_8

><td class="source"><br></td></tr
><tr
id=sl_svn285_9

><td class="source"><br></td></tr
><tr
id=sl_svn285_10

><td class="source">// This kernel computes, per-block, a block-sized scan<br></td></tr
><tr
id=sl_svn285_11

><td class="source">// of the input.  It assumes that the block size evenly<br></td></tr
><tr
id=sl_svn285_12

><td class="source">// divides the input size<br></td></tr
><tr
id=sl_svn285_13

><td class="source">__global__ void inclusive_scan(const unsigned int *input,<br></td></tr
><tr
id=sl_svn285_14

><td class="source">                               unsigned int *result)<br></td></tr
><tr
id=sl_svn285_15

><td class="source">{<br></td></tr
><tr
id=sl_svn285_16

><td class="source">  extern __shared__ unsigned int sdata[];<br></td></tr
><tr
id=sl_svn285_17

><td class="source"><br></td></tr
><tr
id=sl_svn285_18

><td class="source">  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;<br></td></tr
><tr
id=sl_svn285_19

><td class="source"><br></td></tr
><tr
id=sl_svn285_20

><td class="source">  // load input into __shared__ memory<br></td></tr
><tr
id=sl_svn285_21

><td class="source">  unsigned int sum = input[i];<br></td></tr
><tr
id=sl_svn285_22

><td class="source">  sdata[threadIdx.x] = sum;<br></td></tr
><tr
id=sl_svn285_23

><td class="source">  __syncthreads();<br></td></tr
><tr
id=sl_svn285_24

><td class="source">  for(int offset = 1; offset &lt; blockDim.x; offset &lt;&lt;= 1)<br></td></tr
><tr
id=sl_svn285_25

><td class="source">  {<br></td></tr
><tr
id=sl_svn285_26

><td class="source">    if(threadIdx.x &gt;= offset)<br></td></tr
><tr
id=sl_svn285_27

><td class="source">    {<br></td></tr
><tr
id=sl_svn285_28

><td class="source">      sum += sdata[threadIdx.x - offset];<br></td></tr
><tr
id=sl_svn285_29

><td class="source">    }<br></td></tr
><tr
id=sl_svn285_30

><td class="source"><br></td></tr
><tr
id=sl_svn285_31

><td class="source">    // wait until every thread has updated its partial sum<br></td></tr
><tr
id=sl_svn285_32

><td class="source">    __syncthreads();<br></td></tr
><tr
id=sl_svn285_33

><td class="source"><br></td></tr
><tr
id=sl_svn285_34

><td class="source">    // write my partial sum<br></td></tr
><tr
id=sl_svn285_35

><td class="source">    sdata[threadIdx.x] = sum;<br></td></tr
><tr
id=sl_svn285_36

><td class="source"><br></td></tr
><tr
id=sl_svn285_37

><td class="source">    // wait until every thread has written its partial sum<br></td></tr
><tr
id=sl_svn285_38

><td class="source">    __syncthreads();<br></td></tr
><tr
id=sl_svn285_39

><td class="source">  }<br></td></tr
><tr
id=sl_svn285_40

><td class="source"><br></td></tr
><tr
id=sl_svn285_41

><td class="source">  // we&#39;re done! each thread writes out its result<br></td></tr
><tr
id=sl_svn285_42

><td class="source">  result[i] = sdata[threadIdx.x];<br></td></tr
><tr
id=sl_svn285_43

><td class="source">}<br></td></tr
><tr
id=sl_svn285_44

><td class="source">                               <br></td></tr
><tr
id=sl_svn285_45

><td class="source"><br></td></tr
><tr
id=sl_svn285_46

><td class="source">int main(void)<br></td></tr
><tr
id=sl_svn285_47

><td class="source">{<br></td></tr
><tr
id=sl_svn285_48

><td class="source">  // use small input sizes for illustrative purposes<br></td></tr
><tr
id=sl_svn285_49

><td class="source">  const int num_blocks = 4;<br></td></tr
><tr
id=sl_svn285_50

><td class="source">  const int block_size = 16;<br></td></tr
><tr
id=sl_svn285_51

><td class="source">  const int num_elements = num_blocks * block_size;<br></td></tr
><tr
id=sl_svn285_52

><td class="source"><br></td></tr
><tr
id=sl_svn285_53

><td class="source">  // generate random input in [0,5] on the host<br></td></tr
><tr
id=sl_svn285_54

><td class="source">  std::vector&lt;unsigned int&gt; h_input(num_elements);<br></td></tr
><tr
id=sl_svn285_55

><td class="source">  for(unsigned int i = 0; i &lt; num_elements; ++i)<br></td></tr
><tr
id=sl_svn285_56

><td class="source">  {<br></td></tr
><tr
id=sl_svn285_57

><td class="source">    h_input[i] = rand() % 6;<br></td></tr
><tr
id=sl_svn285_58

><td class="source">  }<br></td></tr
><tr
id=sl_svn285_59

><td class="source"><br></td></tr
><tr
id=sl_svn285_60

><td class="source">  // copy input to device memory<br></td></tr
><tr
id=sl_svn285_61

><td class="source">  unsigned int *d_input = 0;<br></td></tr
><tr
id=sl_svn285_62

><td class="source">  hipMalloc((void**)&amp;d_input, sizeof(unsigned int) * num_elements);<br></td></tr
><tr
id=sl_svn285_63

><td class="source">  hipMemcpy(d_input, &amp;h_input[0], sizeof(unsigned int) * num_elements, hipMemcpyHostToDevice);<br></td></tr
><tr
id=sl_svn285_64

><td class="source"><br></td></tr
><tr
id=sl_svn285_65

><td class="source">  // allocate space for the result<br></td></tr
><tr
id=sl_svn285_66

><td class="source">  unsigned int *d_result = 0;<br></td></tr
><tr
id=sl_svn285_67

><td class="source">  hipMalloc((void**)&amp;d_result, sizeof(unsigned int) * num_elements);<br></td></tr
><tr
id=sl_svn285_68

><td class="source"><br></td></tr
><tr
id=sl_svn285_69

><td class="source">  inclusive_scan&lt;&lt;&lt;num_blocks, block_size, block_size * sizeof(unsigned int)&gt;&gt;&gt;(d_input, d_result);<br></td></tr
><tr
id=sl_svn285_70

><td class="source"><br></td></tr
><tr
id=sl_svn285_71

><td class="source">  // copy result to host memory<br></td></tr
><tr
id=sl_svn285_72

><td class="source">  std::vector&lt;unsigned int&gt; h_result(num_elements);<br></td></tr
><tr
id=sl_svn285_73

><td class="source">  hipMemcpy(&amp;h_result[0], d_result, sizeof(unsigned int) * num_elements, hipMemcpyDeviceToHost);<br></td></tr
><tr
id=sl_svn285_74

><td class="source"><br></td></tr
><tr
id=sl_svn285_75

><td class="source">  // print out the results<br></td></tr
><tr
id=sl_svn285_76

><td class="source">  for(int b = 0; b &lt; num_blocks; ++b)<br></td></tr
><tr
id=sl_svn285_77

><td class="source">  {<br></td></tr
><tr
id=sl_svn285_78

><td class="source">    std::cout &lt;&lt; &quot;Block &quot; &lt;&lt; b &lt;&lt; std::endl &lt;&lt; std::endl;<br></td></tr
><tr
id=sl_svn285_79

><td class="source"><br></td></tr
><tr
id=sl_svn285_80

><td class="source">    std::cout &lt;&lt; &quot;Input: &quot; &lt;&lt; std::endl;<br></td></tr
><tr
id=sl_svn285_81

><td class="source">    for(int i = 0; i &lt; block_size; ++i)<br></td></tr
><tr
id=sl_svn285_82

><td class="source">    {<br></td></tr
><tr
id=sl_svn285_83

><td class="source">      printf(&quot;%2d &quot;, h_input[b * block_size + i]);<br></td></tr
><tr
id=sl_svn285_84

><td class="source">    }<br></td></tr
><tr
id=sl_svn285_85

><td class="source">    std::cout &lt;&lt; std::endl;<br></td></tr
><tr
id=sl_svn285_86

><td class="source"><br></td></tr
><tr
id=sl_svn285_87

><td class="source">    std::cout &lt;&lt; &quot;Result: &quot; &lt;&lt; std::endl;<br></td></tr
><tr
id=sl_svn285_88

><td class="source">    for(int i = 0; i &lt; block_size; ++i)<br></td></tr
><tr
id=sl_svn285_89

><td class="source">    {<br></td></tr
><tr
id=sl_svn285_90

><td class="source">      printf(&quot;%2d &quot;, h_result[b * block_size + i]);<br></td></tr
><tr
id=sl_svn285_91

><td class="source">    }<br></td></tr
><tr
id=sl_svn285_92

><td class="source">    std::cout &lt;&lt; std::endl &lt;&lt; std::endl &lt;&lt; std::endl;<br></td></tr
><tr
id=sl_svn285_93

><td class="source">  }<br></td></tr
><tr
id=sl_svn285_94

><td class="source"><br></td></tr
><tr
id=sl_svn285_95

><td class="source">  return 0;<br></td></tr
><tr
id=sl_svn285_96

><td class="source">}<br></td></tr
><tr
id=sl_svn285_97

><td class="source"><br></td></tr
></table></pre>
<pre><table width="100%"><tr class="cursor_stop cursor_hidden"><td></td></tr></table></pre>
</td>
</tr></table>

 
<script type="text/javascript">
 var lineNumUnderMouse = -1;
 
 function gutterOver(num) {
 gutterOut();
 var newTR = document.getElementById('gr_svn285_' + num);
 if (newTR) {
 newTR.className = 'undermouse';
 }
 lineNumUnderMouse = num;
 }
 function gutterOut() {
 if (lineNumUnderMouse != -1) {
 var oldTR = document.getElementById(
 'gr_svn285_' + lineNumUnderMouse);
 if (oldTR) {
 oldTR.className = '';
 }
 lineNumUnderMouse = -1;
 }
 }
 var numsGenState = {table_base_id: 'nums_table_'};
 var srcGenState = {table_base_id: 'src_table_'};
 var alignerRunning = false;
 var startOver = false;
 function setLineNumberHeights() {
 if (alignerRunning) {
 startOver = true;
 return;
 }
 numsGenState.chunk_id = 0;
 numsGenState.table = document.getElementById('nums_table_0');
 numsGenState.row_num = 0;
 if (!numsGenState.table) {
 return; // Silently exit if no file is present.
 }
 srcGenState.chunk_id = 0;
 srcGenState.table = document.getElementById('src_table_0');
 srcGenState.row_num = 0;
 alignerRunning = true;
 continueToSetLineNumberHeights();
 }
 function rowGenerator(genState) {
 if (genState.row_num < genState.table.rows.length) {
 var currentRow = genState.table.rows[genState.row_num];
 genState.row_num++;
 return currentRow;
 }
 var newTable = document.getElementById(
 genState.table_base_id + (genState.chunk_id + 1));
 if (newTable) {
 genState.chunk_id++;
 genState.row_num = 0;
 genState.table = newTable;
 return genState.table.rows[0];
 }
 return null;
 }
 var MAX_ROWS_PER_PASS = 1000;
 function continueToSetLineNumberHeights() {
 var rowsInThisPass = 0;
 var numRow = 1;
 var srcRow = 1;
 while (numRow && srcRow && rowsInThisPass < MAX_ROWS_PER_PASS) {
 numRow = rowGenerator(numsGenState);
 srcRow = rowGenerator(srcGenState);
 rowsInThisPass++;
 if (numRow && srcRow) {
 if (numRow.offsetHeight != srcRow.offsetHeight) {
 numRow.firstChild.style.height = srcRow.offsetHeight + 'px';
 }
 }
 }
 if (rowsInThisPass >= MAX_ROWS_PER_PASS) {
 setTimeout(continueToSetLineNumberHeights, 10);
 } else {
 alignerRunning = false;
 if (startOver) {
 startOver = false;
 setTimeout(setLineNumberHeights, 500);
 }
 }
 }
 function initLineNumberHeights() {
 // Do 2 complete passes, because there can be races
 // between this code and prettify.
 startOver = true;
 setTimeout(setLineNumberHeights, 250);
 window.onresize = setLineNumberHeights;
 }
 initLineNumberHeights();
</script>

 
 
 <div id="log">
 <div style="text-align:right">
 <a class="ifCollapse" href="#" onclick="_toggleMeta(this); return false">Show details</a>
 <a class="ifExpand" href="#" onclick="_toggleMeta(this); return false">Hide details</a>
 </div>
 <div class="ifExpand">
 
 
 <div class="pmeta_bubble_bg" style="border:1px solid white">
 <div class="round4"></div>
 <div class="round2"></div>
 <div class="round1"></div>
 <div class="box-inner">
 <div id="changelog">
 <p>Change log</p>
 <div>
 <a href="/p/stanford-cs193g-sp2010/source/detail?spec=svn285&amp;r=223">r223</a>
 by jaredhoberock
 on Apr 14, 2010
 &nbsp; <a href="/p/stanford-cs193g-sp2010/source/diff?spec=svn285&r=223&amp;format=side&amp;path=/trunk/tutorials/block_scan.cu&amp;old_path=/trunk/tutorials/block_scan.cu&amp;old=">Diff</a>
 </div>
 <pre>Add block scan example code.
</pre>
 </div>
 
 
 
 
 
 
 <script type="text/javascript">
 var detail_url = '/p/stanford-cs193g-sp2010/source/detail?r=223&spec=svn285';
 var publish_url = '/p/stanford-cs193g-sp2010/source/detail?r=223&spec=svn285#publish';
 // describe the paths of this revision in javascript.
 var changed_paths = [];
 var changed_urls = [];
 
 changed_paths.push('/trunk/tutorials/block_scan.cu');
 changed_urls.push('/p/stanford-cs193g-sp2010/source/browse/trunk/tutorials/block_scan.cu?r\x3d223\x26spec\x3dsvn285');
 
 var selected_path = '/trunk/tutorials/block_scan.cu';
 
 
 function getCurrentPageIndex() {
 for (var i = 0; i < changed_paths.length; i++) {
 if (selected_path == changed_paths[i]) {
 return i;
 }
 }
 }
 function getNextPage() {
 var i = getCurrentPageIndex();
 if (i < changed_paths.length - 1) {
 return changed_urls[i + 1];
 }
 return null;
 }
 function getPreviousPage() {
 var i = getCurrentPageIndex();
 if (i > 0) {
 return changed_urls[i - 1];
 }
 return null;
 }
 function gotoNextPage() {
 var page = getNextPage();
 if (!page) {
 page = detail_url;
 }
 window.location = page;
 }
 function gotoPreviousPage() {
 var page = getPreviousPage();
 if (!page) {
 page = detail_url;
 }
 window.location = page;
 }
 function gotoDetailPage() {
 window.location = detail_url;
 }
 function gotoPublishPage() {
 window.location = publish_url;
 }
</script>

 
 <style type="text/css">
 #review_nav {
 border-top: 3px solid white;
 padding-top: 6px;
 margin-top: 1em;
 }
 #review_nav td {
 vertical-align: middle;
 }
 #review_nav select {
 margin: .5em 0;
 }
 </style>
 <div id="review_nav">
 <table><tr><td>Go to:&nbsp;</td><td>
 <select name="files_in_rev" onchange="window.location=this.value">
 
 <option value="/p/stanford-cs193g-sp2010/source/browse/trunk/tutorials/block_scan.cu?r=223&amp;spec=svn285"
 selected="selected"
 >/trunk/tutorials/block_scan.cu</option>
 
 </select>
 </td></tr></table>
 
 
 



 
 </div>
 
 
 </div>
 <div class="round1"></div>
 <div class="round2"></div>
 <div class="round4"></div>
 </div>
 <div class="pmeta_bubble_bg" style="border:1px solid white">
 <div class="round4"></div>
 <div class="round2"></div>
 <div class="round1"></div>
 <div class="box-inner">
 <div id="older_bubble">
 <p>Older revisions</p>
 
 <a href="/p/stanford-cs193g-sp2010/source/list?path=/trunk/tutorials/block_scan.cu&start=223">All revisions of this file</a>
 </div>
 </div>
 <div class="round1"></div>
 <div class="round2"></div>
 <div class="round4"></div>
 </div>
 
 <div class="pmeta_bubble_bg" style="border:1px solid white">
 <div class="round4"></div>
 <div class="round2"></div>
 <div class="round1"></div>
 <div class="box-inner">
 <div id="fileinfo_bubble">
 <p>File info</p>
 
 <div>Size: 2767 bytes,
 97 lines</div>
 
 <div><a href="//stanford-cs193g-sp2010.googlecode.com/svn/trunk/tutorials/block_scan.cu">View raw file</a></div>
 </div>
 
 </div>
 <div class="round1"></div>
 <div class="round2"></div>
 <div class="round4"></div>
 </div>
 </div>
 </div>


</div>

</div>
</div>


<script src="https://ssl.gstatic.com/codesite/ph/18376132045800511552/js/source_file_scripts.js"></script>

 <script type="text/javascript" src="https://ssl.gstatic.com/codesite/ph/18376132045800511552/js/kibbles.js"></script>
 <script type="text/javascript">
 var lastStop = null;
 var initialized = false;
 
 function updateCursor(next, prev) {
 if (prev && prev.element) {
 prev.element.className = 'cursor_stop cursor_hidden';
 }
 if (next && next.element) {
 next.element.className = 'cursor_stop cursor';
 lastStop = next.index;
 }
 }
 
 function pubRevealed(data) {
 updateCursorForCell(data.cellId, 'cursor_stop cursor_hidden');
 if (initialized) {
 reloadCursors();
 }
 }
 
 function draftRevealed(data) {
 updateCursorForCell(data.cellId, 'cursor_stop cursor_hidden');
 if (initialized) {
 reloadCursors();
 }
 }
 
 function draftDestroyed(data) {
 updateCursorForCell(data.cellId, 'nocursor');
 if (initialized) {
 reloadCursors();
 }
 }
 function reloadCursors() {
 kibbles.skipper.reset();
 loadCursors();
 if (lastStop != null) {
 kibbles.skipper.setCurrentStop(lastStop);
 }
 }
 // possibly the simplest way to insert any newly added comments
 // is to update the class of the corresponding cursor row,
 // then refresh the entire list of rows.
 function updateCursorForCell(cellId, className) {
 var cell = document.getElementById(cellId);
 // we have to go two rows back to find the cursor location
 var row = getPreviousElement(cell.parentNode);
 row.className = className;
 }
 // returns the previous element, ignores text nodes.
 function getPreviousElement(e) {
 var element = e.previousSibling;
 if (element.nodeType == 3) {
 element = element.previousSibling;
 }
 if (element && element.tagName) {
 return element;
 }
 }
 function loadCursors() {
 // register our elements with skipper
 var elements = CR_getElements('*', 'cursor_stop');
 var len = elements.length;
 for (var i = 0; i < len; i++) {
 var element = elements[i]; 
 element.className = 'cursor_stop cursor_hidden';
 kibbles.skipper.append(element);
 }
 }
 function toggleComments() {
 CR_toggleCommentDisplay();
 reloadCursors();
 }
 function keysOnLoadHandler() {
 // setup skipper
 kibbles.skipper.addStopListener(
 kibbles.skipper.LISTENER_TYPE.PRE, updateCursor);
 // Set the 'offset' option to return the middle of the client area
 // an option can be a static value, or a callback
 kibbles.skipper.setOption('padding_top', 50);
 // Set the 'offset' option to return the middle of the client area
 // an option can be a static value, or a callback
 kibbles.skipper.setOption('padding_bottom', 100);
 // Register our keys
 kibbles.skipper.addFwdKey("n");
 kibbles.skipper.addRevKey("p");
 kibbles.keys.addKeyPressListener(
 'u', function() { window.location = detail_url; });
 kibbles.keys.addKeyPressListener(
 'r', function() { window.location = detail_url + '#publish'; });
 
 kibbles.keys.addKeyPressListener('j', gotoNextPage);
 kibbles.keys.addKeyPressListener('k', gotoPreviousPage);
 
 
 }
 </script>
<script src="https://ssl.gstatic.com/codesite/ph/18376132045800511552/js/code_review_scripts.js"></script>
<script type="text/javascript">
 function showPublishInstructions() {
 var element = document.getElementById('review_instr');
 if (element) {
 element.className = 'opened';
 }
 }
 var codereviews;
 function revsOnLoadHandler() {
 // register our source container with the commenting code
 var paths = {'svn285': '/trunk/tutorials/block_scan.cu'}
 codereviews = CR_controller.setup(
 {"domainName":null,"assetHostPath":"https://ssl.gstatic.com/codesite/ph","profileUrl":"/u/116699586124044253698/","token":"MBDwIWyWnVeH7W7374m1cEh0iI0:1371808304867","relativeBaseUrl":"","projectName":"stanford-cs193g-sp2010","loggedInUserEmail":"gaurav.sachin007@gmail.com","assetVersionPath":"https://ssl.gstatic.com/codesite/ph/18376132045800511552","projectHomeUrl":"/p/stanford-cs193g-sp2010"}, '', 'svn285', paths,
 CR_BrowseIntegrationFactory);
 
 codereviews.registerActivityListener(CR_ActivityType.REVEAL_DRAFT_PLATE, showPublishInstructions);
 
 codereviews.registerActivityListener(CR_ActivityType.REVEAL_PUB_PLATE, pubRevealed);
 codereviews.registerActivityListener(CR_ActivityType.REVEAL_DRAFT_PLATE, draftRevealed);
 codereviews.registerActivityListener(CR_ActivityType.DISCARD_DRAFT_COMMENT, draftDestroyed);
 
 
 
 
 
 
 
 var initialized = true;
 reloadCursors();
 }
 window.onload = function() {keysOnLoadHandler(); revsOnLoadHandler();};

</script>
<script type="text/javascript" src="https://ssl.gstatic.com/codesite/ph/18376132045800511552/js/dit_scripts.js"></script>

 
 
 
 <script type="text/javascript" src="https://ssl.gstatic.com/codesite/ph/18376132045800511552/js/ph_core.js"></script>
 
 
 
 
</div> 

<div id="footer" dir="ltr">
 <div class="text">
 <a href="/projecthosting/terms.html">Terms</a> -
 <a href="http://www.google.com/privacy.html">Privacy</a> -
 <a href="/p/support/">Project Hosting Help</a>
 </div>
</div>
 <div class="hostedBy" style="margin-top: -20px;">
 <span style="vertical-align: top;">Powered by <a href="http://code.google.com/projecthosting/">Google Project Hosting</a></span>
 </div>

 
 


 
 </body>
</html>

